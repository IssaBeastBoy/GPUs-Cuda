#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
/* Example code taken from "Cheng J. et al. Professional CUDA C Programming"
 * An example of using shared memory to transpose square thread coordinates
 * of a CUDA grid into a global memory array. Different kernels below
 * demonstrate performing reads and writes with different ordering, as well as
 * optimizing using memory padding.
 */

#define BDIMX 4
#define BDIMY 4
#define IPAD  1

void printData(char *msg, int *in,  const int size)
{
    printf("%s: ", msg);

    for (int i = 0; i < size; i++)
    {
        printf("%5d", in[i]);
        fflush(stdout);
    }

    printf("\n");
    return;
}

__global__ void setRowReadRow (int *out)
{
    // static shared memory
    __shared__ int tile[BDIMY][BDIMX];

    // mapping from thread index to global memory index
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    // shared memory store operation
    tile[threadIdx.y][threadIdx.x] = idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[idx] = tile[threadIdx.y][threadIdx.x] ;
}

__global__ void setColReadCol (int *out)
{
    // static shared memory
    __shared__ int tile[BDIMX][BDIMY];

    // mapping from thread index to global memory index
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    // shared memory store operation
    tile[threadIdx.x][threadIdx.y] = idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[idx] = tile[threadIdx.x][threadIdx.y];
}

__global__ void setRowReadCol(int *out)
{
    // static shared memory
    __shared__ int tile[BDIMY][BDIMX];

    // mapping from thread index to global memory index
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    // shared memory store operation
    tile[threadIdx.y][threadIdx.x] = idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[idx] = tile[threadIdx.x][threadIdx.y];
}



__global__ void setRowReadColPad(int *out)
{
    // static shared memory
    __shared__ int tile[BDIMY][BDIMX + IPAD];

    // mapping from thread index to global memory offset
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    // shared memory store operation
    tile[threadIdx.y][threadIdx.x] = idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[idx] = tile[threadIdx.x][threadIdx.y];
}


int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    checkCudaErrors(hipSetDevice(dev));

    hipSharedMemConfig pConfig;
    checkCudaErrors(hipDeviceGetSharedMemConfig ( &pConfig ));
    printf("with Bank Mode:%s ", pConfig == 1 ? "4-Byte" : "8-Byte");

    // set up array size 2048
    int nx = BDIMX;
    int ny = BDIMY;

    bool iprintf = 1;

    if (argc > 1) iprintf = atoi(argv[1]);

    size_t nBytes = nx * ny * sizeof(int);

    // execution configuration
    dim3 block (BDIMX, BDIMY);
    dim3 grid  (1, 1);
    printf("<<< grid (%d,%d) block (%d,%d)>>>\n", grid.x, grid.y, block.x,
           block.y);

    // allocate device memory
    int *d_C;
    checkCudaErrors(hipMalloc((int**)&d_C, nBytes));
    int *gpuRef  = (int *)malloc(nBytes);

    checkCudaErrors(hipMemset(d_C, 0, nBytes));
    setColReadCol<<<grid, block>>>(d_C);
    checkCudaErrors(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("set col read col   ", gpuRef, nx * ny);

    checkCudaErrors(hipMemset(d_C, 0, nBytes));
    setRowReadRow<<<grid, block>>>(d_C);
    checkCudaErrors(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("set row read row   ", gpuRef, nx * ny);

    checkCudaErrors(hipMemset(d_C, 0, nBytes));
    setRowReadCol<<<grid, block>>>(d_C);
    checkCudaErrors(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("set row read col   ", gpuRef, nx * ny);

    checkCudaErrors(hipMemset(d_C, 0, nBytes));
    setRowReadColPad<<<grid, block>>>(d_C);
    checkCudaErrors(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("set row read col pad", gpuRef, nx * ny);

    // free host and device memory
    checkCudaErrors(hipFree(d_C));
    free(gpuRef);

    // reset device
    checkCudaErrors(hipDeviceReset());
    return EXIT_SUCCESS;
}